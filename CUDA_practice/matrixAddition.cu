#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>


__global__ void add(int n, float* x, float* y){

    for(int i = 0; i < n; i++){
        y[i] = x[i] + y[i];
    }

}




int main(){
    int N = 1<<20; // 1M elements

    float *x, float *y;

    // allocate unified memory - accessible from cpu or gpu
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));


    // initialize x and y arrays on the host
    for(int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }


    // Run kernel on 1M elements on the CPU
    add<<<1, 1>>>(N,x,y);

    // wait for GPU to finnish before accessing on host
    hipDeviceSynchronize();

    // free memory
    hipFree(x);
    hipFree(y);

}
