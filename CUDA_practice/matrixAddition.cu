
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__ void add(int n, float* x, float* y){

    int index = threadIdx.x;
    int stride = blockDim.x;

    for(int i = index; i < n; i += stride){
        y[i] = x[i] + y[i];
    }

    //if(index < n){
    //    y[index] = x[index] + y[index];
    //}

}




int main(){
    int N = 1<<20; // 1M elements

    float *x;
    float *y;

    // allocate unified memory - accessible from cpu or gpu
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));


    // initialize x and y arrays on the host
    for(int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }


    // Run kernel on 1M elements on the CPU
    add<<<1, 256>>>(N,x,y);

    // wait for GPU to finnish before accessing on host
    hipDeviceSynchronize();

    // free memory
    hipFree(x);
    hipFree(y);


}
